#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "../include/pagerank.h"

#define H2D (hipMemcpyHostToDevice)
#define D2H (hipMemcpyDeviceToHost)

int MONTE_CARLO = 0;

__global__ void compute(const int num_active_nodes, int* active_nodes, float* value, float* new_value, const int* rowdeg, const int* colptr, const int* row, const int* col)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < num_active_nodes) {
        int n = active_nodes[tid];
        new_value[n] = 1 - alpha;

        for (int e = colptr[n]; e < colptr[n + 1]; e++) {
            new_value[n] += alpha * value[row[e]] / (float)rowdeg[row[e]];
        }
    }
}

__global__ void find_active(const int num_active_nodes, int* active_nodes, float* value, float* new_value, int* is_next_nodes)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < num_active_nodes) {
        int n = active_nodes[tid];
        is_next_nodes[tid] = abs(value[n] - new_value[n]) > epsilon? 1: 0;
    }
}

__global__ void copy_value(const int num_active_nodes, int* active_nodes, float* value, float* new_value)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < num_active_nodes) {
        int n = active_nodes[tid];
        value[n] = new_value[n];
    }
}

__global__ void coalesce_next_active(const int num_active_nodes, int* active_nodes, int* next_nodes, int* is_next_nodes)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < num_active_nodes && is_next_nodes[tid] < is_next_nodes[tid + 1]) {
        next_nodes[is_next_nodes[tid]] = active_nodes[tid];
    }
}

__global__ void copy_active(const int num_active_nodes, int* active_nodes, int* next_nodes)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < num_active_nodes) {
        active_nodes[tid] = next_nodes[tid];
    }
}

void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* colptr, const int* row, const int* col)
{
    float *d_value, *d_new_value;
    int *d_active_nodes, *d_is_next_nodes, *d_next_nodes, *d_rowdeg, *d_colptr, *d_row, *d_col;

    int num_active_nodes = nodes;
    int* active_nodes = new int[nodes];
    for (int n = 0; n < nodes; n++) {
        active_nodes[n] = n;
    }

    const int threads_per_block = 128;

    hipMalloc(&d_value, sizeof(float) * nodes);
    hipMalloc(&d_new_value, sizeof(float) * nodes);

    hipMalloc(&d_active_nodes, sizeof(int) * nodes);
    hipMalloc(&d_is_next_nodes, sizeof(int) * (nodes + 1));
    hipMalloc(&d_next_nodes, sizeof(int) * nodes);

    hipMalloc(&d_rowdeg, sizeof(int) * nodes);
    hipMalloc(&d_colptr, sizeof(int) * (nodes + 1));
    hipMalloc(&d_row, sizeof(int) * edges);
    hipMalloc(&d_col, sizeof(int) * edges);

    hipMemcpy(d_value, value, sizeof(float) * nodes, H2D);
    hipMemcpy(d_active_nodes, active_nodes, sizeof(int) * nodes, H2D);

    hipMemcpy(d_rowdeg, rowdeg, sizeof(int) * nodes, H2D);
    hipMemcpy(d_colptr, colptr, sizeof(int) * (nodes + 1), H2D);
    hipMemcpy(d_row, row, sizeof(int) * edges, H2D);
    hipMemcpy(d_col, col, sizeof(int) * edges, H2D);

    while (true) {
        compute<<<num_active_nodes/threads_per_block+1,threads_per_block>>>(num_active_nodes, d_active_nodes, d_value, d_new_value, d_rowdeg, d_colptr, d_row, d_col);
        find_active<<<num_active_nodes/threads_per_block+1,threads_per_block>>>(num_active_nodes, d_active_nodes, d_value, d_new_value, d_is_next_nodes);
        copy_value<<<num_active_nodes/threads_per_block+1,threads_per_block>>>(num_active_nodes, d_active_nodes, d_value, d_new_value);

        thrust::exclusive_scan(thrust::device, d_is_next_nodes, d_is_next_nodes + num_active_nodes + 1, d_is_next_nodes);
        coalesce_next_active<<<num_active_nodes/threads_per_block+1, threads_per_block>>>(num_active_nodes, d_active_nodes, d_next_nodes, d_is_next_nodes);

        hipMemcpy(&num_active_nodes, &d_is_next_nodes[num_active_nodes], sizeof(int), D2H);

        if (num_active_nodes == 0)
            break;
        
        copy_active<<<num_active_nodes/threads_per_block+1,threads_per_block>>>(num_active_nodes, d_active_nodes, d_next_nodes);
    }

    hipMemcpy(value, d_value, sizeof(float) * nodes, D2H);

    hipFree(d_value);
    hipFree(d_new_value);

    hipFree(d_active_nodes);
    hipFree(d_is_next_nodes);
    hipFree(d_next_nodes);

    hipFree(d_rowdeg);
    hipFree(d_colptr);
    hipFree(d_row);
    hipFree(d_col);
}
