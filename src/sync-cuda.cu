#include "hip/hip_runtime.h"
#include "include/pagerank.h"

#define H2D (hipMemcpyHostToDevice)
#define D2H (hipMemcpyDeviceToHost)

__global__ void compute(const int nodes, const int edges, float* value, float* new_value, const int* rowdeg, const int* colptr, const int* row, const int* col) 
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        new_value[tid] = 1 - alpha;

        for (int e = colptr[tid]; e < colptr[tid + 1]; e++) {
            new_value[tid] += alpha * value[row[e]] / (float)rowdeg[row[e]];
        }
    }

}

__global__ void copy_value(const int nodes, float* value, float* new_value)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        value[tid] = new_value[tid];
    }
}

void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* colptr, const int* row, const int* col)
{
    float *d_value, *d_new_value;
    int *d_rowdeg, *d_colptr, *d_row, *d_col;

    hipMalloc(&d_value, sizeof(float) * nodes);
    hipMalloc(&d_new_value, sizeof(float) * nodes);
    hipMalloc(&d_rowdeg, sizeof(int) * nodes);
    hipMalloc(&d_colptr, sizeof(int) * (nodes + 1));
    hipMalloc(&d_row, sizeof(int) * edges);
    hipMalloc(&d_col, sizeof(int) * edges);

    hipMemcpy(d_value, value, sizeof(float) * nodes, H2D);
    hipMemcpy(d_rowdeg, rowdeg, sizeof(int) * nodes, H2D);
    hipMemcpy(d_colptr, colptr, sizeof(int) * (nodes + 1), H2D);
    hipMemcpy(d_row, row, sizeof(int) * edges, H2D);
    hipMemcpy(d_col, col, sizeof(int) * edges, H2D);

    for (int i = 0; i < iteration; i++) {
        compute<<<nodes/512+1, 512>>>(nodes, edges, d_value, d_new_value, d_rowdeg, d_colptr, d_row, d_col);
        copy_value<<<nodes/512+1, 512>>>(nodes, d_value, d_new_value);
    }

    hipMemcpy(value, d_value, sizeof(float) * nodes, D2H);

    hipFree(d_value);
    hipFree(d_new_value);
    hipFree(d_rowdeg);
    hipFree(d_colptr);
    hipFree(d_row);
    hipFree(d_col);
}