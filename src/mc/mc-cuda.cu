#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "../include/pagerank.h"

#define H2D (hipMemcpyHostToDevice)
#define D2H (hipMemcpyDeviceToHost)

#define WARP_SIZE 32

int MONTE_CARLO = 1;

__global__ void setup(const int nodes, float* value, hiprandStateMRG32k3a *state)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < nodes) {
        value[tid] = 0;
    }
    if (threadIdx.x < WARP_SIZE) {
        int rid = threadIdx.x + blockIdx.x * WARP_SIZE;
        hiprand_init(0, rid, 0, &state[rid]);
    }
}

__global__ void random_walk(const int nodes, float* value, const int* rowptr, const int* col, hiprandStateMRG32k3a *state)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int rid = (threadIdx.x % WARP_SIZE) + blockIdx.x * WARP_SIZE;

    if (tid < nodes) {
        int cur = tid;
        for (int i = 0; i < length; i++) {
            int deg = rowptr[cur + 1] - rowptr[cur];
            if (hiprand_uniform(&state[rid]) < alpha)
                cur = deg == 0? cur: col[ rowptr[cur] + (int)(hiprand_uniform(&state[rid]) * deg) ];
            else
                cur = tid;

            atomicAdd(&value[cur], 1);
        }
    }
}

__global__ void normalize(const int nodes, float* value)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        value[tid] /= length;
    }
}


void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* rowptr, const int* row, const int* col) {
    float *d_value;
    int *d_rowptr, *d_col;
    hiprandStateMRG32k3a *state;

    const int threads_per_block = 512;

    hipMalloc(&state, sizeof(hiprandStateMRG32k3a) * (nodes / threads_per_block + 1) * WARP_SIZE);

    hipMalloc(&d_value, sizeof(float) * nodes);

    hipMalloc(&d_rowptr, sizeof(int) * (nodes + 1));
    hipMalloc(&d_col, sizeof(int) * edges);

    hipMemcpy(d_rowptr, rowptr, sizeof(int) * (nodes + 1), H2D);
    hipMemcpy(d_col, col, sizeof(int) * edges, H2D);

    setup<<<nodes/threads_per_block+1, threads_per_block>>>(nodes, d_value, state);
    random_walk<<<nodes/threads_per_block+1, threads_per_block>>>(nodes, d_value, d_rowptr, d_col, state);
    normalize<<<nodes/threads_per_block+1, threads_per_block>>>(nodes, d_value);

    hipMemcpy(value, d_value, sizeof(float) * nodes, D2H);

    hipFree(state);
    hipFree(d_value);
    hipFree(d_rowptr);
    hipFree(d_col);
}
