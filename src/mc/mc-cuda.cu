#include "hip/hip_runtime.h"
#include "include/pagerank.h"

#include <hiprand/hiprand_kernel.h>

#define H2D (hipMemcpyHostToDevice)
#define D2H (hipMemcpyDeviceToHost)

int MONTE_CARLO = 1;

__global__ void setup_rand(hiprandStateMRG32k3a *state)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(0, tid, 0, &state[tid]);
}

__global__ void random_walk(const int nodes, float* value, const int* rowptr, const int* row, const int* col, hiprandStateMRG32k3a *state)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        int cur = tid;
        for (int i = 0; i < length; i++) {
            if (hiprand_uniform(&state[tid]) < alpha)
                cur = col[rowptr[cur] + (int)(hiprand_uniform(&state[tid]) * rowdeg[cur])];
            else
                cur = tid;

            atomicAdd(&value[cur], 1);
        }
    }
}

__global___ void normalize(const int nodes, float* value)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        value[cur] /= length;
    }
}


void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* rowptr, const int* row, const int* col) {
    float *d_value;
    int *d_rowptr, *d_row, *d_col;
    hiprandStateMRG32k3a *state;

    hipMalloc(&state, sizeof(hiprandStateMRG32k3a) * nodes)

    hipMalloc(&d_value, sizeof(float) * nodes);

    hipMalloc(&d_rowptr, sizeof(int) * (nodes + 1));
    hipMalloc(&d_row, sizeof(int) * nodes);
    hipMalloc(&d_col, sizeof(int) * nodes);

    hipMemset(&d_value, 0, sizeof(float) * nodes);

    hipMemcpy(d_rowptr, rowptr, sizeof(int) * (nodes + 1), H2D);
    hipMemcpy(d_row, row, sizeof(int) * nodes, H2D);
    hipMemcpy(d_col, col, sizeof(int) * nodes, H2D);

    const int threads_per_block = 128;

    setup_rand<<<nodes/threads_per_block+1, threads_per_block>>>(state);
    random_walk<<<nodes/threads_per_block+1, threads_per_block>>>(nodes, d_value, d_rowptr, d_row, d_col, state);
    normalize<<<nodes/threads_per_block+1, threads_per_block>>>(nodes, d_value);

    hipMemcpy(value, d_value, sizeof(float) * nodes, D2H);

    hipFree(state);
    hipFree(d_value);
    hipFree(d_rowptr);
    hipFree(d_row);
    hipFree(d_col);
}