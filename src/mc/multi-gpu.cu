#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include "../include/pagerank.h"

#define H2D (hipMemcpyHostToDevice)
#define D2H (hipMemcpyDeviceToHost)

#define WARP_SIZE 32

int MONTE_CARLO = 1;

__global__ void setup(const int nodes, float* value, hiprandStateMRG32k3a *state)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < nodes) {
        value[tid] = 0;
    }
    if (threadIdx.x < WARP_SIZE) {
        int rid = threadIdx.x + blockIdx.x * WARP_SIZE;
        hiprand_init(0, rid, 0, &state[rid]);
    }
}

__global__ void random_walk(const int nodes, float* value, const int* rowptr, const int* col, hiprandStateMRG32k3a *state)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int rid = (threadIdx.x % WARP_SIZE) + blockIdx.x * WARP_SIZE;

    if (tid < nodes) {
        int cur = tid;
        for (int i = 0; i < length; i++) {
            int deg = rowptr[cur + 1] - rowptr[cur];
            if (hiprand_uniform(&state[rid]) < alpha)
                cur = deg == 0? cur: col[ rowptr[cur] + (int)(hiprand_uniform(&state[rid]) * deg) ];
            else
                cur = tid;

            atomicAdd(&value[cur], 1);
        }
    }
}

__global__ void normalize(const int nodes, float* value)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        value[tid] /= length;
    }
}

void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* rowptr, const int* row, const int* col) {
    int ngpus;
    hipGetDeviceCount(&ngpus);

    float **d_value = new float*[ngpus];
    //float **h_value = new float*[ngpus];
    hipStream_t* streams = new hipStream_t[ngpus];

    const int threads_per_block = 512;

    for (int i = 0; i < ngpus; i++) {
        hipSetDevice(i);
        
        hipStreamCreate(&streams[i]);
    
        int *d_rowptr, *d_col;
        hiprandStateMRG32k3a *state;

        hipMalloc(&state, sizeof(hiprandStateMRG32k3a) * (nodes / threads_per_block + 1) * WARP_SIZE);

        hipMalloc(&d_value[i], sizeof(float) * nodes);
        //h_value[i] = new float[nodes];

        hipMalloc(&d_rowptr, sizeof(int) * (nodes + 1));
        hipMalloc(&d_col, sizeof(int) * edges);

        hipMemcpyAsync(d_rowptr, rowptr, sizeof(int) * (nodes + 1), H2D, streams[i]);
        hipMemcpyAsync(d_col, col, sizeof(int) * edges, H2D, streams[i]);

        setup<<<nodes/threads_per_block+1, threads_per_block, 0, streams[i]>>>(nodes, d_value[i], state);
        random_walk<<<nodes/threads_per_block+1, threads_per_block, 0, streams[i]>>>(nodes, d_value[i], d_rowptr, d_col, state);
        normalize<<<nodes/threads_per_block+1, threads_per_block, 0, streams[i]>>>(nodes, d_value[i]);

        //hipMemcpyAsync(h_value[i], d_value[i], sizeof(float) * nodes, D2H, streams[i]);
        
        //hipFree(d_value[i]);
        hipFree(state);
        hipFree(d_rowptr);
        hipFree(d_col);
    }

    /*
    #pragma omp parallel for
    for (int i = 0; i < nodes; i++) {
        value[i] = 0;
        for (int j = 0; j < ngpus; j++) {
            value[i] += h_value[j][i] / ngpus;
        }
    }
    */

    hipStream_t* streams = new hipStream_t[ngpus];
    for ()
}

