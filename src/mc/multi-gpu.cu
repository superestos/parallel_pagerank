#include "hip/hip_runtime.h"
/*
void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* rowptr, const int* row, const int* col) {

    int ngpus;
    hipGetDeviceCount(&ngpus);

    float **d_value = new float*[ngpus];
    hipStream_t* streams = new hipStream_t[ngpus];

    const int threads_per_block = 1024;

    omp_set_num_threads(ngpus);
    #pragma omp parallel
    {
        int i = omp_get_thread_num();
        hipSetDevice(i);
        
        hipStreamCreate(&streams[i]);
        
        int *d_rowptr, *d_col;
        hiprandStateMRG32k3a *state;

        hipMalloc(&state, sizeof(hiprandStateMRG32k3a) * nodes);

        hipMalloc(&d_value[i], sizeof(float) * nodes);

        hipMalloc(&d_rowptr, sizeof(int) * (nodes + 1));
        hipMalloc(&d_col, sizeof(int) * edges);

        setup<<<nodes/threads_per_block+1, threads_per_block, 0, streams[i]>>>(nodes, d_value[i], state);
        random_walk<<<nodes/threads_per_block+1, threads_per_block, 0, streams[i]>>>(nodes, d_value[i], d_rowptr, d_col, state);
        normalize<<<nodes/threads_per_block+1, threads_per_block, 0, streams[i]>>>(nodes, d_value[i]);

        hipMemcpyAsync(d_rowptr, rowptr, sizeof(int) * (nodes + 1), H2D, streams[i]);
        hipMemcpyAsync(d_col, col, sizeof(int) * edges, H2D, streams[i]);

        hipDeviceSynchronize();

        hipFree(d_rowptr);
        hipFree(d_col);
        hipFree(state);
    }


    /*
    hipStream_t *streams = new hipStream_t[ngpus];

    float **d_value = new float*[ngpus];
    int **d_rowptr = new int*[ngpus];
    int **d_col = new int*[ngpus];
    hiprandStateMRG32k3a **state = new hiprandStateMRG32k3a *[ngpus];
    
}

*/

#include <hiprand/hiprand_kernel.h>

#include "../include/pagerank.h"

#define H2D (hipMemcpyHostToDevice)
#define D2H (hipMemcpyDeviceToHost)

#define WARP_SIZE 32

int MONTE_CARLO = 1;

__global__ void setup(const int nodes, float* value, hiprandStateMRG32k3a *state)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < nodes) {
        value[tid] = 0;
    }
    if (threadIdx.x < WARP_SIZE) {
        int rid = threadIdx.x + blockIdx.x * WARP_SIZE;
        hiprand_init(0, rid, 0, &state[rid]);
    }
}

__global__ void random_walk(const int nodes, float* value, const int* rowptr, const int* col, hiprandStateMRG32k3a *state)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int rid = (threadIdx.x % WARP_SIZE) + blockIdx.x * WARP_SIZE;

    if (tid < nodes) {
        int cur = tid;
        for (int i = 0; i < length; i++) {
            int deg = rowptr[cur + 1] - rowptr[cur];
            if (hiprand_uniform(&state[rid]) < alpha)
                cur = deg == 0? cur: col[ rowptr[cur] + (int)(hiprand_uniform(&state[rid]) * deg) ];
            else
                cur = tid;

            atomicAdd(&value[cur], 1);
        }
    }
}

__global__ void normalize(const int nodes, float* value)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < nodes) {
        value[tid] /= length;
    }
}


void pagerank(const int nodes, const int edges, float* value, const int* rowdeg, const int* rowptr, const int* row, const int* col) {
    int *d_rowptr, *d_col;
    hiprandStateMRG32k3a *state;

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMalloc(&state, sizeof(hiprandStateMRG32k3a) * (nodes / threads_per_block + 1) * WARP_SIZE);

    hipMalloc(&d_value, sizeof(float) * nodes);

    hipMalloc(&d_rowptr, sizeof(int) * (nodes + 1));
    hipMalloc(&d_col, sizeof(int) * edges);

    hipMemcpyAsync(d_rowptr, rowptr, sizeof(int) * (nodes + 1), H2D, stream);
    hipMemcpyAsync(d_col, col, sizeof(int) * edges, H2D, stream);

    setup<<<nodes/threads_per_block+1, threads_per_block, 0, stream>>>(nodes, d_value, state);
    random_walk<<<nodes/threads_per_block+1, threads_per_block, 0, stream>>>(nodes, d_value, d_rowptr, d_col, state);
    normalize<<<nodes/threads_per_block+1, threads_per_block, 0, stream>>>(nodes, d_value);

    hipMemcpyAsync(value, d_value, sizeof(float) * nodes, D2H, stream);

    hipFree(state);
    hipFree(d_value);
    hipFree(d_rowptr);
    hipFree(d_col);
}
